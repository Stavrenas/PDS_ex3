#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h> // sqrt, M_PI
#include <stdbool.h>
#include <time.h>
#include <sys/time.h>
#include "utilities.h"
#include "cudaUtilities.h"

struct timeval tic()
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv;
}

double toc(struct timeval begin)
{
    struct timeval end;
    gettimeofday(&end, NULL);
    double stime = ((double)(end.tv_sec - begin.tv_sec) * 1000) +
                   ((double)(end.tv_usec - begin.tv_usec) / 1000);
    stime = stime / 1000;
    return (stime);
}

float gaussian(float sigma, float x)
{
    return (1 / (sigma * sqrt(2 * M_PI))) * exp(-x * x / (2 * sigma * sigma));
}

int *readCSV(int *n, char *file) //n represents total number of pixels
{
    FILE *matFile;
    matFile = fopen(file, "r");
    if (matFile == NULL)
    {
        printf("Could not open file %s\n", file);
        exit(-1);
    }
    int pixels, error;
    pixels = 1;
    error = 1;
    int *array = (int *)malloc(pixels * sizeof(int));
    while (error)
    {
        error = fscanf(matFile, "%d,", &array[pixels - 1]);
        if (error != 1)
        {
            printf("Finished reading image \n");
            *n = sqrt(pixels);
            fclose(matFile);
            return array;
        }
        pixels++;
        array = (int *)realloc(array, pixels * sizeof(int));
    }
    *n = sqrt(pixels);

    fclose(matFile);
    return array;
}

float *normalizeImage(int *image, int size) //size represents the dimension
{
    int max = 0;
    for (int i = 0; i < size * size; i++)
    {
        if (image[i] > max)
            max = image[i];
    }
    float *array = (float *)malloc(size * size * sizeof(float));
    for (int i = 0; i < size * size; i++)
    {
        array[i] = ((float)image[i]) / max;
    }
    printf("Finished normalizing\n");
    return array;
}

float *addNoiseToImage(float *image, int size)
{
    srand(time(NULL));
    float *noisy = (float *)malloc(size * size * sizeof(float));

    float random_value, effect;
    for (int i = 0; i < size * size; i++)
    {
        random_value = ((float)rand() / RAND_MAX * 20 - 10);
        effect = gaussian(2, random_value) - 0.05;
        noisy[i] = (effect + 1) * image[i]; //add gaussian noise
        if (noisy[i] < 0)
            noisy[i] = 0;
        else if (noisy[i] > 1)
            noisy[i] = 1;
    }
    printf("Finished adding noise\n");
    return noisy;
}

void writeToCSV(float *image, int size, char *name)
{
    FILE *filepointer;
    char *filename = (char *)malloc((strlen(name) + 4) * sizeof(char));
    sprintf(filename, "%s.csv", name);
    filepointer = fopen(filename, "w"); //create a file
    for (int i = 0; i < size; i++)
    {
        for (int j = 0; j < size; j++)
            fprintf(filepointer, "%f,", image[i * size + j]); //write each pixel value

        fprintf(filepointer, "\n");
    }
}

float findMax(float *array, int size)
{
    float max = 0;
    for (int i = 0; i < size; i++)
    {
        if (array[i] > max)
            max = array[i];
    }
    return max;
}

float *createPatchesRowMajor(float *image, int size, int patchSize)
{
    //We assume that patchSize is an odd number//
    //In order to create the patches we must consider that the pixels are stored in Row-Major format//
    //A simple aproach is to handle the patches also in the same format//
    int patchLimit = (patchSize - 1) / 2;
    int patchIterator, imageIterator;
    float *patches = (float *)malloc(size * size * patchSize * patchSize * sizeof(float));

    for (int i = 0; i < size; i++)
    {
        for (int j = 0; j < size; j++) //go to each pixel of the image
        {
            float *patch = (float *)malloc(patchSize * patchSize * sizeof(float)); //We assume that (i,j) is the pixel on the centre
            for (int k = -patchLimit; k <= patchLimit; k++)
            {
                for (int m = -patchLimit; m <= patchLimit; m++) //go to each pixel of the patch: i*size +j
                {
                    patchIterator = (k + patchLimit) * patchSize + (m + patchLimit);
                    imageIterator = (i + k) * size + (j + m);
                    patch[patchIterator] = -1;

                    if (imageIterator >= 0 && imageIterator < size * size) //filter out of image pixels
                    {

                        if (!(j < patchLimit && m < -j) && !(j >= size - patchLimit && m >= size - j))
                            //!(j  < patchLimit && m +  < 0) filters pixels that are on the left side of the patch
                            //!(j  >= size - patchLimit && m  >=size - j ) filters pixels that are on the right side of the patch
                            patch[patchIterator] = image[imageIterator];
                    }
                }
            }
            for (int o = 0; o < patchSize * patchSize; o++)
                patches[i*size*patchSize*patchSize + j*patchSize*patchSize + o] = patch[o];
            free(patch);
        }
    }
    return patches;
}

void printPatch(float *patch, int patchSize)
{

    for (int i = 0; i < patchSize; i++)
    {
        for (int j = 0; j < patchSize; j++)
        {
            if (patch[i * patchSize + j] == -1)
                printf("    x    ");
            else
                printf("%f ", patch[i * patchSize + j]);
        }
        printf("\n");
    }
}

float *findRemoved(float *noisy, float *denoised, int size)
{
    int totalPixels = size * size;
    float *removed = (float *)malloc(totalPixels * sizeof(float));
    for (int i = 0; i < totalPixels; i++)
        removed[i] = denoised[i] - noisy[i];
    printf("Finished finding removed\n");
    return removed;
}

// __device__ float getPatchElement(float *image, int size, int pixel, int position, int patchSize)
// {
//     //returns the element in a certain patch position
//     //without a need to save patch in memory

//     int patchLimit = (patchSize - 1) / 2;
//     float result = -1;
//     int j = pixel % size;                                                                    //int i = pixel / size;
//     int m = position % patchSize - patchLimit;                                               //int k = position / patchSize - patchLimit;
//     int imageIterator = (pixel / size + position / patchSize - patchLimit) * size + (j + m); //int imageIterator = (i + k) * size + (j + m);

//     if (imageIterator >= 0 && imageIterator < size * size) //filter out of image pixels
//     {
//         if (!(j < patchLimit && m < -j) && !(j >= size - patchLimit && m >= size - j))
//             result = image[imageIterator];
//     }
//     return result;
// }

// __global__ void distanceSquaredCuda(int size, float *x, float *y, float *z)
// {
//     int i = blockIdx.x * blockDim.x + threadIdx.x;
//     if (i < size)
//         z[i] = (x[i] - y[i]) * (x[i] - y[i]);
// }

// __global__ void gaussianDistanceCuda(int size, float *distances, float *gaussianWeights, int patchSize, float *x)
// {
//     int i = blockIdx.x * blockDim.x + threadIdx.x;
//     if (i < size)
//     {
//         int patchLimit = (patchSize - 1) / 2;
//         int m = i % patchSize - patchLimit;
//         int k = i / patchSize - patchLimit;
//         int distance = m * m + k * k;
//         x[i] *= gaussianWeights[distance];
//     }
// }


float *calculateDistances(float *patches, int size, int patchSize, float *gaussianWeights, float sigmaDist)
{
    int totalPixels = size * size;
    int patchLimit = (patchSize - 1) / 2;
    float *cudaPatches, *cudaGaussianWeights, *cudaDistances;
    hipMalloc(&cudaPatches, totalPixels * patchSize * patchSize* sizeof(float));
    hipMalloc(&cudaDistances, totalPixels * totalPixels * sizeof(float));
    hipMalloc(&cudaGaussianWeights, (patchSize + patchLimit) * sizeof(float));
    hipMemcpy(cudaGaussianWeights, gaussianWeights, (patchSize + patchLimit) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(cudaPatches, patches, totalPixels * patchSize * patchSize * sizeof(float), hipMemcpyHostToDevice);

    findPatchDistances<<<size, size>>>(cudaPatches, size, patchSize, cudaGaussianWeights, cudaDistances, sigmaDist);

    float *distances = (float *)malloc(totalPixels * totalPixels * sizeof(float));
    hipMemcpy(distances, cudaDistances, (totalPixels * totalPixels * sizeof(float)), hipMemcpyDeviceToHost);
    hipFree(cudaPatches);
    hipFree(cudaDistances);
    hipFree(cudaGaussianWeights);
    return distances;
}

__global__ void findPatchDistances(float *patches, int size, int patchSize, float *gaussianWeights, float *distances, float sigmaDist)
{

    int col = threadIdx.x; //0.. size-1
    int row = blockIdx.x;  //0.. size-1
    int totalPixels = size * size;
    int totalPatch=patchSize*patchSize;
    //printf("i is %d j is %d\n",i,j);

    for (int i = row*size; i < (row+1)*size; i++)
    {
        for (int j = col*size; j < (col+1)*size; j++)
        {
            int patchLimit = (patchSize - 1) / 2;
            float result, sum = 0;

            for (int k = -patchLimit; k <= patchLimit; k++)
            {
                for (int m = -patchLimit; m <= patchLimit; m++) //go to each pixel of the patch: i*size +j
                {
                    int patchIterator = (k + patchLimit) * patchSize + (m + patchLimit);
                    if (patches[i * totalPatch + patchIterator] != -1 && patches[j * totalPatch + patchIterator] != -1) //this means out of bounds
                    {
                        int distance = m * m + k * k; 
                        result = (patches[i * totalPatch + patchIterator] - patches[j * totalPatch + patchIterator]) *
                            (patches[i * totalPatch + patchIterator] - patches[j * totalPatch + patchIterator]) * gaussianWeights[distance];
                        sum += result;
                    }
                }
            }
            distances[i*totalPixels + j] = exp(-sum / (sigmaDist * sigmaDist));
        }
    }
}
